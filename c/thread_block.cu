
#include <hip/hip_runtime.h>
#include <stdio.h>

__device__ int getGlobalIdx()
{
	return blockIdx.x * blockDim.x * blockDim.y + threadIdx.y * blockDim.x + threadIdx.x;
}

__global__ void kernel(int * d_in)
{
	int global_idx = getGlobalIdx();
	printf("Hello world! I'm a thread %d in block %d, my global id is %d and my value is %d\n", threadIdx.x, blockIdx.x, global_idx, d_in[global_idx]);
}

int main(int argc, char** argv)
{
	int array_size = 8;
	int array_bytes = array_size * sizeof(float);
        
	// generate the input array on the host
	int h_in[array_size];
        for(int i = 0; i < array_size; i++)
	{
	  h_in[i] = i;
        }

        //declare GPU memory pointers
        int * d_in;
        
        //allocate GPU memory
	hipMalloc((void**) &d_in, array_bytes);

        //transfer the array to the GPU
        hipMemcpy(d_in, h_in, array_bytes, hipMemcpyHostToDevice);
	
        //launch the kernel
        kernel<<<1, array_size>>>(d_in);
        
        //force the printf()s to flush
        hipDeviceSynchronize();
        
        hipFree(d_in);
       
        return 0;        
}

