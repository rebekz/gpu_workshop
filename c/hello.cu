
#include <hip/hip_runtime.h>
#include <stdio.h>

#define NUM_BLOCKS 16
#define BLOCK_WIDTH 1

__global__ void hello()
{
    printf("Hello world! I'm a thread in block %d\n", blockIdx.x);
}

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess) 
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}

int main(int argc,char **argv)
{
    // launch the kernel
    hello<<<NUM_BLOCKS, BLOCK_WIDTH>>>();

    // force the printf()s to flush
   gpuErrchk(hipDeviceSynchronize());

    printf("That's all!\n");

    return 0;
}
