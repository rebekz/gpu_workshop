
#include <hip/hip_runtime.h>
#include <stdio.h>

__device__ int getGlobalIdx()
{
	return blockIdx.x * blockDim.x * blockDim.y + threadIdx.y * blockDim.x + threadIdx.x;
}

__global__ void kernel(int * d_in, int * d_out)
{
	int global_idx = getGlobalIdx();
	printf("Hello world! I'm a thread %d in block %d, my global id is %d and my value is %d\n", threadIdx.x, blockIdx.x, global_idx, d_in[global_idx]);
	d_out[global_idx] = d_in[global_idx] * d_in[global_idx];
}

int main(int argc, char** argv)
{
	int array_size = 8;
	int array_bytes = array_size * sizeof(float);
        
	// generate the input array on the host
	int h_in[array_size];
	printf("input: \n");
        for(int i = 0; i < array_size; i++)
	{
	  h_in[i] = i;
	  printf("%d ", h_in[i]);
        }
        int h_out[array_size];
	printf("\n");        

        //declare GPU memory pointers
        int * d_in;
        int * d_out;
        
        //allocate GPU memory
	hipMalloc((void**) &d_in, array_bytes);
	hipMalloc((void**) &d_out, array_bytes);

        //transfer the array to the GPU
        hipMemcpy(d_in, h_in, array_bytes, hipMemcpyHostToDevice);
	
        //launch the kernel
        kernel<<<1, array_size>>>(d_in, d_out);
        
        //force the printf()s to flush
        hipDeviceSynchronize();
        
	//copy back the result array to the CPU
	hipMemcpy(h_out, d_out, array_bytes, hipMemcpyDeviceToHost);

	//print out the resulting array
        printf("Result:\n");
	for(int i = 0; i < array_size; i++)
	{
		printf("%d ", h_out[i]);
	}
	
	printf("\n");

        hipFree(d_in);
	hipFree(d_out);
       
        return 0;        
}

